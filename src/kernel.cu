#include "hip/hip_runtime.h"
#include <cstdio>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <vector>

#include "Mesh.hpp"
#include "SimpleTimer.hpp"

//TODO: throw exception outside and use spdlog there, otherwise problems wit compilation

static std::string to_string(hipError_t error) {
	char buf[256];
	snprintf(buf, 256, "%d", error);
	return buf;
}

class CudaError : public std::runtime_error {
public:
	CudaError(std::string source, hipError_t errorCode) :
		std::runtime_error(source + ": code" + to_string(errorCode) + ": " + hipGetErrorString(errorCode)) {}
};

#define checkCudaErrors(val) checkError((val), #val, __FILE__, __LINE__)
void checkError(hipError_t result, const char* calledFunc, const char* file, int line) {
	if (result) {
		std::ostringstream ss;
		ss << file << ": " << line << " {" << calledFunc << '}';

		throw CudaError(ss.str(), result);
	}
}

__global__ void meshUpdateKernel(float* mesh_in, float* mesh_out, size_t pitch, unsigned size) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x > 0 && x < size - 1 && y > 0 && y < size - 1) {
		const float t_left = *getElem(mesh_in, pitch, y, x - 1);
		const float t_right = *getElem(mesh_in, pitch, y, x + 1);
		const float t_top = *getElem(mesh_in, pitch, y - 1, x);
		const float t_bottom = *getElem(mesh_in, pitch, y + 1, x);

		const float newTemperature = (t_left + t_right + t_top + t_bottom) / 4;

		*getElem(mesh_out, pitch, y, x) = newTemperature;
	}
}

std::vector<std::vector<float>> cuda_heat_compute(int blockDimX, int blockDimY, int meshSize, int steps) {
	meshSize += 2; // add edge rows/cols resembling environment temperature

	size_t pitch;
	float* temperature = allocMeshLinear(pitch, meshSize);
	size_t d_pitch;
	float *d_temperature_in, *d_temperature_out;

	checkCudaErrors(hipMallocPitch(&d_temperature_in, &d_pitch, meshSize * sizeof(float), meshSize));
	checkCudaErrors(hipMallocPitch(&d_temperature_out, &d_pitch, meshSize * sizeof(float), meshSize));

	{
		SimpleTimer t("CUDA computations");
		dim3 blockSize(blockDimX, blockDimY);
		unsigned computedGridDimX = (meshSize + blockSize.x - 1) / blockSize.x;
		unsigned computedGridDimY = (meshSize + blockSize.y - 1) / blockSize.y;
		dim3 gridSize(computedGridDimX, computedGridDimY);

		checkCudaErrors(
			hipMemcpy2D(d_temperature_in, d_pitch, temperature, pitch, meshSize * sizeof(float), meshSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy2D(
			d_temperature_out, d_pitch, d_temperature_in, d_pitch, meshSize * sizeof(float), meshSize, hipMemcpyDeviceToDevice));

		for (int step = 0; step < steps; ++step) {
			meshUpdateKernel<<<gridSize, blockSize>>>(d_temperature_in, d_temperature_out, d_pitch, meshSize);
			checkCudaErrors(hipGetLastError()); // Check for any errors launching the kernel
			checkCudaErrors(
				hipDeviceSynchronize()); // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
			std::swap(d_temperature_in, d_temperature_out);
		}

		checkCudaErrors(
			hipMemcpy2D(temperature, pitch, d_temperature_in, d_pitch, meshSize * sizeof(float), meshSize, hipMemcpyDeviceToHost));
	}

	SimpleTimer t("Computation results processing");
	std::vector<std::vector<float>> result;
	for (int y = 1; y < meshSize - 1; ++y) {
		result.emplace_back();
		for (int x = 1; x < meshSize - 1; ++x) {
			result.back().emplace_back(*getElem(temperature, pitch, x, y));
		}
	}

	if (!validateResults(temperature, pitch)) { return {}; }

	//TODO: RAII guard to avoid resource leaks
	delete[] temperature;
	checkCudaErrors(hipFree(d_temperature_in));
	checkCudaErrors(hipFree(d_temperature_out));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	checkCudaErrors(hipDeviceReset());

	return result;
}